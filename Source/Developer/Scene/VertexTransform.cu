#include "hip/hip_runtime.h"
#include "Cuda/CudaAssert.h"
#include "VertexTransform.h"

#include "TensorOps/Implementation/HelperMacros.h"
namespace LTSE::Graphics
{

#define THREADS_PER_BLOCK 512

    namespace Kernels
    {
        CUDA_KERNEL_DEFINITION void SkinnedVertexTransform( VertexData *aOutTransformedVertices, VertexData *aVertices, math::mat4 *aObjectToWorldTransform,
                                                            math::mat4 *aJointMatrices, uint32_t aJointCount, uint32_t aObjectCount, uint32_t *aObjectOffsets,
                                                            uint32_t *aObjectVertexCount )
        {
            uint32_t lObjectOffset      = aObjectOffsets[blockIdx.x];
            uint32_t lObjectVertexCount = aObjectVertexCount[blockIdx.x];

            uint32_t lVertexID = blockIdx.y * LTSE::TensorOps::Private::ThreadsPerBlock + threadIdx.x;

            RETURN_UNLESS( lVertexID < lObjectVertexCount );

            math::mat4 lTransform = aObjectToWorldTransform[blockIdx.x];

            VertexData lVertex = aVertices[lObjectOffset + lVertexID];

            math::mat4 lSkinTransform = lVertex.Weights.x * aJointMatrices[int( lVertex.Bones.x )] + lVertex.Weights.y * aJointMatrices[int( lVertex.Bones.y )] +
                                        lVertex.Weights.z * aJointMatrices[int( lVertex.Bones.z )] + lVertex.Weights.w * aJointMatrices[int( lVertex.Bones.w )];

            math::mat4 lFinalTransform = lTransform * lSkinTransform;

            aOutTransformedVertices[lObjectOffset + lVertexID] = lVertex;

            aOutTransformedVertices[lObjectOffset + lVertexID].Position = math::vec3( lFinalTransform * math::vec4( lVertex.Position, 1.0f ) );
            aOutTransformedVertices[lObjectOffset + lVertexID].Normal   = normalize( transpose( inverse( mat3( lFinalTransform ) ) ) * lVertex.Normal );
        }

        CUDA_KERNEL_DEFINITION void StaticVertexTransform( VertexData *aOutTransformedVertices, VertexData *aVertices, math::mat4 *aObjectToWorldTransform, uint32_t aObjectCount,
                                                           uint32_t *aObjectOffsets, uint32_t *aObjectVertexCount )
        {
            uint32_t lObjectOffset      = aObjectOffsets[blockIdx.x];
            uint32_t lObjectVertexCount = aObjectVertexCount[blockIdx.x];

            uint32_t lVertexID = blockIdx.y * LTSE::TensorOps::Private::ThreadsPerBlock + threadIdx.x;

            RETURN_UNLESS( lVertexID < lObjectVertexCount );

            math::mat4 lTransform = aObjectToWorldTransform[blockIdx.x];
            VertexData lVertex    = aVertices[lObjectOffset + lVertexID];

            aOutTransformedVertices[lObjectOffset + lVertexID] = lVertex;

            aOutTransformedVertices[lObjectOffset + lVertexID].Position = math::vec3( lTransform * math::vec4( lVertex.Position, 1.0f ) );
            aOutTransformedVertices[lObjectOffset + lVertexID].Normal   = normalize( transpose( inverse( mat3( lTransform ) ) ) * lVertex.Normal );
        }
    } // namespace Kernels

    extern "C" __global__ void __kernel__TransformVertices( math::mat4 a_Transform, GPUArray<VertexData> a_Vertices, GPUArray<math::vec3> o_VertexOutput )
    {
        const int l_RayID = blockDim.x * blockIdx.x + threadIdx.x;

        if( l_RayID >= a_Vertices.ElementCount )
            return;

        o_VertexOutput.DevicePointer[l_RayID] = math::vec3( a_Transform * math::vec4( a_Vertices.DevicePointer[l_RayID].Position, 1.0f ) );
    }

    void StaticVertexTransform( VertexData *aOutTransformedVertices, VertexData *aVertices, math::mat4 *aObjectToWorldTransform, uint32_t aObjectCount, uint32_t *aObjectOffsets,
                                uint32_t *aObjectVertexCount, uint32_t aMaxVertexCount )
    {
        int lBlockCount = ( aMaxVertexCount / LTSE::TensorOps::Private::ThreadsPerBlock ) + 1;
        dim3 lGridDim( aObjectCount, lBlockCount, 1 );
        dim3 lBlockDim( LTSE::TensorOps::Private::ThreadsPerBlock );

        Kernels::StaticVertexTransform<<<lGridDim, lBlockDim>>>( aOutTransformedVertices, aVertices, aObjectToWorldTransform, aObjectCount, aObjectOffsets, aObjectVertexCount );
    }

    void SkinnedVertexTransform( VertexData *aOutTransformedVertices, VertexData *aVertices, math::mat4 *aObjectToWorldTransform, math::mat4 *aJointMatrices, uint32_t aJointCount,
                                 uint32_t aObjectCount, uint32_t *aObjectOffsets, uint32_t *aObjectVertexCount, uint32_t aMaxVertexCount )
    {
        int lBlockCount = ( aMaxVertexCount / LTSE::TensorOps::Private::ThreadsPerBlock ) + 1;
        dim3 lGridDim( aObjectCount, lBlockCount, 1 );
        dim3 lBlockDim( LTSE::TensorOps::Private::ThreadsPerBlock );

        Kernels::SkinnedVertexTransform<<<lGridDim, lBlockDim>>>( aOutTransformedVertices, aVertices, aObjectToWorldTransform, aJointMatrices, aJointCount, aObjectCount,
                                                                  aObjectOffsets, aObjectVertexCount );
    }

    void TransformVertices( math::mat4 a_Transform, GPUArray<VertexData> a_Vertices, GPUArray<math::vec3> o_VertexOutput )
    {
        int l_GridDimensionX = ( a_Vertices.ElementCount / THREADS_PER_BLOCK ) + 1;

        dim3 l_GridDim( l_GridDimensionX, 1, 1 );
        dim3 l_BlockDim( THREADS_PER_BLOCK, 1 );

        hipMemset( o_VertexOutput.DevicePointer, 0, o_VertexOutput.ElementCount * sizeof( math::vec3 ) );
        __kernel__TransformVertices<<<l_GridDim, l_BlockDim>>>( a_Transform, a_Vertices, o_VertexOutput );
        hipDeviceSynchronize();
    }

} // namespace LTSE::Graphics